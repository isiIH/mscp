#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <fstream>
#include <sstream>
#include <chrono>
#include "include/BasicCDS.h"
#include <cmath>
#include <set>
#include <map>
#include <unordered_map>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cds;

#define PRINT 1
#define CHECK 0

#define BS 1024

typedef struct{
	int value;
	int rep;
	vector<int> subSets;
} item;

// Structure with all globals parameters program
typedef struct {
    int search;
    int k = 0;
	ulong* X;
	vector<vector<int>> F;
    set<int> chi;
    map<int,int> elem_pos;
    vector<ulong*> bF;
    vector<item> mp;
    vector<ulong*> unique_elements;
    vector<ulong*> greedy_sol;
    vector<ulong*> exh_sol;
    ulong sizeF, sizeNF;
	ulong n, m, nWX;
    int nt;
} ParProg;

ParProg* par;

void readFile(string filename);
void readFileScp(string filename);
void readFilePartition(string filename);
void analizeF();
void preprocess();

void linearSearch();
void binarySearch(int l, int r);
void exponentialSearch();
void reverseSearch();

void exhaustive_sol();
void greedy();

__device__ int coefBin(int n, int k);
__global__ void generateCombinationsKernel(int m, int k, ulong* d_combinations, ulong* X, ulong* sol, int nWX, ulong combCount, bool* found);
__device__ bool isCovered(ulong* chosenSets, int k, ulong* X, ulong nWX, ulong* sharedMem);
bool launchKernel(int k);

int countSet(const ulong* S);
int intersectionLength(const ulong* A, const ulong* B);

void printSubset(const ulong *S);
void printSubsets(const vector<ulong*> &C);

int main(int argc, char** argv) {

    if(argc !=3){
		cout << "./opt <filename> <search>" << endl;
		exit(EXIT_FAILURE);
	}

    par = new ParProg();
    par->search = atoi(argv[2]);
    if(par->search < 0 || par->search > 3){
        cout << "Invalid Search Type!\n0: Secuential Search\n1: Binary Search\n2: Exponential Search\n3: Reverse Search" << endl;
        exit(EXIT_FAILURE);
    }

    readFile(argv[1]);
    auto start_time = chrono::high_resolution_clock::now();
    analizeF();
    auto end_time = chrono::high_resolution_clock::now();
    auto dur_analyze = chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();


    cout  << "X: " << par->n
        << " | F: " << par->m << endl;

    par->sizeF = par->m*sizeof(ulong)*par->n;
    par->sizeNF = par->m*sizeof(ulong)*par->nWX;

	cout << "nWX = " << par->nWX << endl;
	cout << " size for F[] = " << par->sizeF/(1024.0*1024.0) << " MiB" << endl;
	cout << " size for nF[] = " << par->sizeNF/(1024.0*1024.0) << " MiB" << endl;

    if(CHECK) {
        for(vector<int> set : par->F) {
            for(int val : set) {
                cout << val << " - ";
            }
            cout << endl;
        }

        printSubsets(par->bF);
    }

    //PREPROCESS
    start_time = chrono::high_resolution_clock::now();
    preprocess();
    end_time = chrono::high_resolution_clock::now();
    auto dur_preprocess = chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();

    //GREEDY
    start_time = chrono::high_resolution_clock::now();
    greedy();
    end_time = chrono::high_resolution_clock::now();
    auto dur_greedyExh = chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();
    dur_greedyExh += dur_preprocess + dur_analyze;

    //NEW EXHAUSTIVE ALGORITHM
    start_time = chrono::high_resolution_clock::now();
    exhaustive_sol();
    end_time = chrono::high_resolution_clock::now();
    auto dur_opt = chrono::duration_cast<chrono::microseconds>(end_time - start_time).count();
    dur_opt += dur_preprocess + dur_analyze;

    if(CHECK) {
        cout << "SOL: " << endl;
        printSubsets(par->exh_sol);
    }
    cout << "------------------------" << endl;
    cout << "Greedy Cardinality: " << par->greedy_sol.size() << endl;
    cout << "Time [s]: " << dur_greedyExh/1000000.0 << endl;
	cout << "Optimal Cardinality: " << par->exh_sol.size() << endl;
    cout << "Time [s]: " << dur_opt/1000000.0 << endl;

    return 0;
}

void readFile(string filename) {
    if (filename.substr(0,3) == "scp") readFileScp(filename);
    else readFilePartition(filename);
}

void readFileScp(string filename) {
    cout << "Reading file " << filename << "..." << endl;
    string nametxt = "test_scp/" + filename;
    ifstream file(nametxt.c_str());
    if(file.fail()){
        cout << "File not found!" << endl;
        exit(EXIT_FAILURE);
    }
    string line,item;
    int i;

    //m & n
	getline(file>>std::ws,line);
    istringstream ss(line);
    ss >> (par->n) >> (par->m);

    //Costs
    i = 0;
    while(i < par->m)
    {
        getline(file>>std::ws,line);
        istringstream iss(line);
        while (getline(iss>>std::ws, item, ' ')){i++;}
    }

    //Sets
    int numCover;
    int j;
    par->F.resize(par->m);
    for(i=0; i<par->n; i++) {
        getline(file>>std::ws,line);
        numCover = stoi(line);

        j = 0;
        while(j < numCover){
            getline(file>>std::ws,line);
            istringstream iss(line);
            while (getline(iss, item, ' ')) {
                par->F[stoi(item)-1].push_back(i+1);
                j++;
            }
        }
    }
    file.close();
}

void readFilePartition(string filename) {
    if(PRINT) cout << "Reading file " << filename << "..." << endl;
    string nametxt = "test_partition/" + filename;
    ifstream file(nametxt.c_str());
    if(file.fail()){
        cout << "File not found!" << endl;
        exit(EXIT_FAILURE);
    }
    string line,item;

    //m & n
	getline(file>>std::ws,line);
    istringstream ss(line);
    ss >> (par->n) >> (par->m);

    //Sets
    vector<int> sub;
    for (int i = 0; i < par->m; i++) {
        getline(file>>std::ws,line);
        istringstream ss(line);
        getline(ss>>std::ws, item, ' ');
        getline(ss>>std::ws, item, ' ');

        while (getline(ss>>std::ws, item, ' ')) {
            sub.push_back(stoi(item));
        }
        (par->F).push_back(sub);
        sub.clear();
    }
    file.close();
}

void analizeF() {
    unordered_map<int, vector<int>> inSet;
    for( int i=0; i<par->F.size(); i++ ) {
        for( int e : par->F[i] ) {
            par->chi.insert(e);
            inSet[e].push_back(i);
        }
    }

    par->n = par->chi.size();

    par->nWX = (par->n)/(sizeof(ulong)*8);
    if ((par->n)%(sizeof(ulong)*8)>0) par->nWX++;
    par->X = new ulong[par->nWX];
    fill(par->X, par->X + par->nWX, 0);

    int pos = 0;
    par->mp = vector<item>(par->n);
    for(pair<int, vector<int>> values : inSet){
        setBit64(par->X, pos);
        par->elem_pos[values.first] = pos;
        par->mp[pos].value = values.first;
        par->mp[pos].subSets = values.second;
        par->mp[pos].rep = values.second.size();
        pos++;
    }

    ulong *bset;
    for( int i=0; i<par->F.size(); i++ ) {
        bset = new ulong[par->nWX];
        fill(bset, bset + par->nWX, 0);

        for( int e : par->F[i] ) {
            setBit64(bset, par->elem_pos[e]);
            inSet[e].push_back(i);
        }

        par->bF.push_back(bset);
    }

    if(CHECK) {
        for(item mp_item : par->mp) {
            cout << "(" << mp_item.value << ") |" << mp_item.rep << "| => ";
            for (int index : mp_item.subSets) {
                cout << index << " ";
            }
            cout << endl;
        }
    }

    sort(par->mp.begin(), par->mp.end(), [&](item a, item b){return a.rep < b.rep;});

    if(CHECK) {
        cout << "Universe elements = " << endl;
        for( pair<int, int> values : par->elem_pos ) if(getBit64(par->X, values.second)) cout << values.first << " ";
        cout << endl;
        cout << "X = " << countSet(par->X) << endl;
        cout << "n = " << par->n << endl;
        cout << "F = " << par->bF.size() << endl;
        cout << "m = " << par->m << endl;

        for(item mp_item : par->mp) {
            cout << " - " << mp_item.value << " - " << endl;
            cout << mp_item.rep << " subsets." << endl;
            // for (int setIndex : mp_item.subSets) printSubset(par->bF[setIndex]);
        }
    }
}

void greedy() {
    int i;
    ulong* U = new ulong[par->nWX];
    for(i=0; i<par->nWX; i++) U[i] = par->X[i];
    vector<ulong*> subsets = par->bF;
    vector<ulong*> C = par->unique_elements;
    int posSet;
    int maxLengthSS = 0;
    int lengthSS;

    while( countSet(U) > 0 ) {

        for( i=0; i<subsets.size(); i++ ){
            lengthSS = intersectionLength(U, subsets[i]);
            if(lengthSS > maxLengthSS) {
                maxLengthSS = lengthSS;
                posSet = i;
            }
        }

        for(i=0; i<par->nWX; i++) U[i] = U[i] & ~subsets[posSet][i];
        C.push_back(subsets[posSet]);
        subsets.erase(subsets.begin()+posSet);

        maxLengthSS = 0;
    }

    par->greedy_sol = C;
}

void checkCudaError(hipError_t result, const char* msg) {
    if (result != hipSuccess) {
        cerr << msg << ": " << hipGetErrorString(result) << endl;
        exit(EXIT_FAILURE);
    }
}

void exhaustive_sol() {
    cout << "-----------------------------------------------------------" << endl;
    cout << "Executing new exhaustive algorithm";
    switch (par->search) {
        case 0: cout << " with sequential search..." << endl; break;
        case 1: cout << " with binary search..." << endl; break;
        case 2: cout << " with exponential search..." << endl; break;
        case 3: cout << " with reverse search..." << endl; break;
    }
    cout << "-----------------------------------------------------------" << endl;

    // Calcular mínimo número de subconjuntos
    vector<ulong*> Fsort = par->bF;
    sort(Fsort.begin(), Fsort.end(), [&](ulong* a, ulong* b){return countSet(a) > countSet(b);});
    int minSS = 0;
    while(minSS < par->n) {
        minSS += countSet(Fsort[par->k]);
        par->k++;
    }

    //Iterar desde K hasta encontrar el óptimo
    int max_k = par->greedy_sol.size() - par->unique_elements.size();
    cout << "Search Range = [" << par->k << " - " << max_k << "]" << endl;

    switch (par->search) {
        case 0: //Búsqueda secuencial
            linearSearch(); break;
        case 1: //Búsqueda binaria
            binarySearch(par->k, max_k); break;
        case 2: //Búsqueda exponencial
            exponentialSearch(); break;
        case 3: //Búsqueda secuencial inversa (greedy--)
            reverseSearch(); break;
    }
    par->exh_sol.insert(par->exh_sol.end(), par->unique_elements.begin(), par->unique_elements.end());
}

bool launchKernel(int k) {
    //Calcular num combinaciones
    ulong combCount = 1;
    for(int i=0; i<k; i++) {
        combCount *= (par->m-i);
        combCount /= (i+1);
    }
    cout << "Comb. count: " << combCount << endl;

    dim3 threadsPerBlock(BS);
    dim3 numBlocks((combCount + BS -1) / BS);
    size_t sharedMemSize = BS * k * par->nWX * sizeof(ulong) + BS * par->nWX * sizeof(ulong);

    cout << "tpb: " << BS << endl;
    cout << "num_blocks: " << (combCount + BS -1) / BS << endl;

    int uSize = par->nWX * sizeof(ulong);
    int fSize = par->m * uSize;
    int solSize = k * uSize;

    cout << "uSize: " << uSize << endl;
    cout << "fSize: " << fSize << endl;
    cout << "solSize: " << solSize << endl;

    // Family of subsets
    ulong* d_comb;
    checkCudaError(hipMalloc(&d_comb, fSize), "Failed to allocate device memory for d_comb");
    for (int i = 0; i < par->m; i++) {
        checkCudaError(hipMemcpy(&d_comb[i * par->nWX], par->bF[i], uSize, hipMemcpyHostToDevice), "Failed to copy data from host to device");
    }

    // Universe
    ulong* d_X;
    checkCudaError(hipMalloc(&d_X, uSize), "Failed to allocate device memory for d_X");
    checkCudaError(hipMemcpy(d_X, par->X, uSize, hipMemcpyHostToDevice), "Failed to copy data from host to device");

    // Solution mscp
    ulong* h_sol = new ulong[k*par->nWX];
    ulong* d_sol;
    checkCudaError(hipMalloc(&d_sol, solSize), "Failed to allocate device memory for d_sol");

    //found
    bool* d_found;
    checkCudaError(hipMalloc(&d_found, sizeof(bool)), "Failed to allocate device memory for d_found");
    checkCudaError(hipMemset(d_found, 0, sizeof(bool)), "Failed to set device memory for d_found");

    generateCombinationsKernel<<<numBlocks, threadsPerBlock, sharedMemSize>>>(par->m, k, d_comb, d_X, d_sol, par->nWX, combCount, d_found);
    checkCudaError(hipDeviceSynchronize(), "Kernel execution failed");

    checkCudaError(hipMemcpy(h_sol, d_sol, solSize, hipMemcpyDeviceToHost), "Failed to copy data from device to host");
    bool h_found;
    checkCudaError(hipMemcpy(&h_found, d_found, sizeof(bool), hipMemcpyDeviceToHost), "Failed to copy found flag from device to host");

    if(h_found) {
        for(int i=0; i<k; i++) {
            ulong* ss = new ulong[par->nWX];
            for(int j=0; j<par->nWX; j++) {
                if(CHECK) printSubset(h_sol);
                ss[j] = h_sol[i*par->nWX+j];
            }
            par->exh_sol.push_back(ss);
        }
    }

    hipFree(d_comb);
    hipFree(d_sol);
    hipFree(d_X);
    hipFree(d_found);

    return h_found;
}

__device__ int coefBin(int n, int k){

    if((2*k) > n) k = n - k;
    int combCount = 1;
    for (int i = 0; i < k; i++) {
        combCount *= (n - i);
        combCount /= (i + 1);
    }
    return combCount;
}

__global__ void generateCombinationsKernel(int m, int k, ulong* d_combinations, ulong* X, ulong* sol, int nWX, ulong combCount, bool* found) {
    extern __shared__ ulong shared_mem[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // tid fuera de rango
    if (tid >= combCount || *found) return;

    int combId = tid;
    int offset = 0;
    ulong* uComb = shared_mem + threadIdx.x * k * nWX; //comb única generada por tid

    // Calcular combinación única por tid
    for (int i = 0; i < k; i++) {
        for (int j = offset; j < m; j++) {
            int countRest = coefBin((m-j-1), (k-i-1)); //num combinaciones restantes luego de agregar 1
            if (combId < countRest) { //valor de j se agrega a la combinación
                for(int l=0; l<nWX; l++) uComb[i*nWX+l] = d_combinations[j*nWX+l];
                offset = j + 1;
                break;
            } else { //j++, disminución de num combinaciones
                combId -= countRest;
            }
        }
    }

    //Verificar si la comb cubre el universo
    if(!*found && isCovered(uComb, k, X, nWX, shared_mem + (blockDim.x * k * nWX) + (threadIdx.x * nWX))) {
        printf("tid = %d found a solution!\n", tid);

        //Copiar comb a sol
        if (atomicCAS((int*)found, 0, 1) == 0) {
            printf("tid = %d copying the uComb!\n", tid);
            // Copy comb to sol
            for (int i = 0; i < k; i++) {
                for (int j = 0; j < nWX; j++) sol[i * nWX + j] = uComb[i * nWX + j];
            }
        }

    }
}

__device__ bool isCovered(ulong* chosenSets, int k, ulong* X, ulong nWX, ulong* sharedMem) {
    // Calcular unión de chosenSets
    ulong* C = sharedMem;
    for (int i = 0; i < nWX; i++) C[i] = 0;
    for(int i=0; i<k; i++) for(int j=0; j<nWX; j++) C[j] |= chosenSets[i*nWX+j];

    // Calcular número de conjuntos cubiertos
    bool isCov = true;
    for (int i = 0; i < nWX; i++) if ((C[i] & X[i]) != X[i]) {
        isCov = false;
        break;
    }
    return isCov;
}

void linearSearch() {
    bool found = false;
    while(!found) {
        if(PRINT) cout << "K = " << par->k << endl;
        auto start = chrono::high_resolution_clock::now();
        found = launchKernel(par->k);
        auto end = chrono::high_resolution_clock::now();

        auto time = std::chrono::duration_cast<chrono::microseconds>(end - start).count();
        if(PRINT) cout << "Time: " << time / 1000000.0 << "[s]" << endl;
        //No se encuentra una solución de tamaño k, aumentamos en 1
        par->k++;
    }
}

void binarySearch(int l, int r) {
    int m = l;
    bool found;
    while(l <= r) {
        if(PRINT) cout << "K = " << m << endl;
        auto start = chrono::high_resolution_clock::now();
        found = launchKernel(m);
        auto end = chrono::high_resolution_clock::now();
        auto time = std::chrono::duration_cast<chrono::microseconds>(end - start).count();
        if(PRINT) cout << "Time: " << time << "[s]" << endl;
        if (found) r = m - 1;
        else l = m + 1;
        m = l + (r - l)/2;
    }
}

void exponentialSearch() {
    int exp = 1;
    int greedySize = par->greedy_sol.size() - par->unique_elements.size();
    bool found = false;

    while(par->k <= greedySize && !found) {
        cout << "K = " << par->k << endl;
        auto start = chrono::high_resolution_clock::now();
        found = launchKernel(par->k);
        auto end = chrono::high_resolution_clock::now();
        auto time = std::chrono::duration_cast<chrono::microseconds>(end - start).count();
        if(PRINT) cout << "Time: " << time / 1000000.0 << "[s]" << endl;
        
        if(!found){
            par->k += exp;
            exp *= 2;
        }
    }

    //Realizar búsqueda binaria en un rango más pequeño
    int l = par->k - exp/2 + 1;
    int r = min(par->k-1, greedySize);
    cout << "Search range for binary search: [" << l << " - " << r << "]" << endl;
    binarySearch(l, r);
}

void reverseSearch() {
    bool found = true;
    int k = par->greedy_sol.size() - par->unique_elements.size();
    while(k >= par->k && found) {
        if(PRINT) cout << "K = " << k << endl;
        auto start = chrono::high_resolution_clock::now();
        found = launchKernel(k);
        auto end = chrono::high_resolution_clock::now();
        auto time = chrono::duration_cast<chrono::microseconds>(end - start).count();
        if(PRINT) cout << "Time: " << time / 1000000.0 << "[s]" << endl;
        //No se encuentra una solución de tamaño k, disminuimos en 1
        k--;
    }
}

void preprocess() {
    cout << "------------------------" << endl;
    cout << "Executing PreSetCover..." << endl;
    cout << "------------------------" << endl;
    // Add uniques elements
    int setIndex;
    ulong* S;
    while(par->mp[0].rep == 1) {
        setIndex = par->mp[0].subSets[0];
        S = par->bF[setIndex];

        // Eliminar subsets del map que no se usen
        for(int e : par->F[setIndex]) {
            par->mp.erase(remove_if(par->mp.begin(), par->mp.end(), [e](const item& mp) {return mp.value == e;}), par->mp.end());
            cleanBit64(par->X,par->elem_pos[e]);
        }
        par->unique_elements.push_back(S);
    }

    for(ulong* S : par->unique_elements) {
        par->bF.erase(find(par->bF.begin(), par->bF.end(), S));
        par->m--;
    }

    cout << "Added " << par->unique_elements.size() << " subsets " << endl; 
    par->n = countSet(par->X);
    cout << "|X| = " << par->n << endl;
    cout << "|F| = " << par->m << endl;
}

int intersectionLength(const ulong* A, const ulong* B) {
    int cont = 0;
    for(int i=0; i<par->nWX; i++) cont += __builtin_popcountl(A[i] & B[i]);
    return cont;
}

int countSet(const ulong* S){
    int cont = 0;
    for(int i=0; i<par->nWX; i++) {
        cont += __builtin_popcountl(S[i]);
    }
    return cont;
}

void printSubset(const ulong *S) {
    for (int i=0; i<par->nWX; i++){
        printBitsUlong(S[i]);
        cout << " - ";
    }
    cout << endl;
}

void printSubsets(const vector<ulong*> &C) {
    for(ulong* S : C) {
        printSubset(S);
    }
}